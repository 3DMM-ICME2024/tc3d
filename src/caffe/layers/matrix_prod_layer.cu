#include <vector>

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/matrix_prod_layer.hpp"
//#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void MatrixProdLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data0 = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const Dtype* bottom_data1 = bottom[1]->gpu_data();
  for (int i=0; i<bottom[1]->shape(0); ++i){
    // B0 (K*M) * B1 (M*N)
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, K_, N_, M_, (Dtype)1.,
        bottom_data0+i*K_*M_, bottom_data1+i*M_*N_, (Dtype)0., top_data+i*K_*N_);
  }
}

template <typename Dtype>
void MatrixProdLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    // Gradient with respect to weight
    for (int i=0; i<bottom[1]->shape(0); ++i){
      // B0^T (M*K) * D (K*N)
      caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, M_, N_, K_, (Dtype)1.,
          bottom_data+i*K_*M_, top_diff+i*K_*N_, (Dtype)0.,
          bottom[1]->mutable_gpu_diff()+i*N_*M_);
    }
  }
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_data = bottom[1]->gpu_data();
    // Gradient with respect to bottom data
    for (int i=0; i<bottom[1]->shape(0); ++i){
      // D (K*N) * B1^T (N*M)
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, K_, M_, N_, (Dtype)1.,
          top_diff+i*N_*K_, bottom_data+i*N_*M_,  (Dtype)0.,
          bottom[0]->mutable_gpu_diff()+i*K_*M_);
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(MatrixProdLayer);

}  // namespace caffe
